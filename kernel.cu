﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "./include/Matrix.hpp"
#include "./include/DeviceMatrix.cuh"
#include "./include/Models/MLP/Layer.cuh"
#include "./include/Models/Model.cuh"
#include "./include/Models/Activations/Sigmoid.cuh"
#include "./include/Models/CostFunctions/MSE.cuh"
#include "./include/Models/TrainingData/MLP_TrainingData.cuh"
#include "./include/Models/CostFunctions/BCE.cuh"
#include "./include/Models/Activations/ReLU.cuh"

using namespace dl;


/*
interface:

copy ctor = deep copy
copy assignment = deep copy
clone = deep copy (return shared_ptr)
move ctor = ownership transfer
move assignment = ownership transfer
shared_ptr<LearningUnit> + shared_ptr<LearningUnit> = shared_ptr<Model> | Model.units = {shared_ptr, shared_ptr}
Model.bind = operator+

training data se moze djeliti (shared_ptr)
Model(vector<shared_ptr<LearningUnit>>) = radi shallow copy, samo pointere uzima na postojece learning unite, ovo omogucuje djeljenje parametara ali zahtjeva vecu odgovornost
svi ostalo konstruktori u pravilu rade deep copy

*/

#include <iostream>



void f() {

	{


		Model model({
			std::make_shared<MLP::Layer>(MLP::Layer::RandomLayer(2, 4)),
			std::make_shared<ReLU>(),
			std::make_shared<MLP::Layer>(MLP::Layer::RandomLayer(4, 6)),
			std::make_shared<ReLU>(),
			std::make_shared<MLP::Layer>(MLP::Layer::RandomLayer(6, 4)),
			std::make_shared<Sigmoid>(),
			std::make_shared<MLP::Layer>(MLP::Layer::RandomLayer(4, 1)),
			std::make_shared<Sigmoid>()
			});


		std::shared_ptr<TrainingData> data = std::make_shared<TrainingData>();

		for (float i = 0; i < 0.5f; i += 0.1f) {
			for (float j = 0; j < 0.5f; j += 0.1f) {

				data->add({ i, j }, { i + j });
			}
		}

		model.setTrainingData(data);
		model.setCostFunction(std::make_shared<BCE>());


		model.trainMiniBatchSGD(2000, 28, 0.05);

		model.evaluate();


		std::cout << "first time : " << DeviceMatrix::instances << std::endl;



	}


}


int main()
{

	//f();
	
	
	for (int i = 1; i < 0; ++i) {


	DeviceMatrix m = DeviceMatrix::Random(100, i);

	DeviceMatrix a = DeviceMatrix::matColReduce(m);
	DeviceMatrix b = DeviceMatrix::matColReduceV2(m);

	if (a.downloadToHost() != b.downloadToHost()) {
		printf("everithings wrong");
	}

	printf("case %d is good\n", i);
	
	
}

	/*

	Model model({
		std::make_shared<MLP::Layer>(MLP::Layer::RandomLayer(2, 500)),
		std::make_shared<Sigmoid>(),
		std::make_shared<MLP::Layer>(MLP::Layer::RandomLayer(500, 300)),
		std::make_shared<Sigmoid>(),
		std::make_shared<MLP::Layer>(MLP::Layer::RandomLayer(300, 15)),
		std::make_shared<Sigmoid>(),
		std::make_shared<MLP::Layer>(MLP::Layer::RandomLayer(15, 1)),
		std::make_shared<Sigmoid>()
		});

	std::shared_ptr<TrainingData> data = std::make_shared<TrainingData>();


	for (float x = 0.0; x < 0.5; x += 0.01) {
		for (float y = 0.0; y < 0.5; y += 0.01) {
			data->add({ x, y }, { x + y });
		}
	}

	

	model.setTrainingData(data);
	model.setCostFunction(std::make_shared<MSE>());

	model.trainSingleBatchGD(10000, 0.2f);

	model.print();

	model.evaluate();*/



    return 0;
}

