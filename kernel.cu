﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "./include/Matrix.hpp"
#include "./include/DeviceMatrix.cuh"
#include "./include/Models/MLP/Layer.cuh"
#include "./include/Models/Model.cuh"

using namespace dl;


class Base {
public:
	int x;
};

class Derived : public Base { public: int y;  };

int main()
{




	MLP::Layer layer = MLP::Layer::RandomLayer(20, 20);
	

	layer.setInput(DeviceMatrix::Random(20, 1, { -10.0f, 1.0f }));

	MLP::Layer layer2(DeviceMatrix::Random(20, 20), DeviceMatrix::Random(20, 1));
	
	layer2.setInput(layer.forward());

	

	std::unique_ptr<Model> model = std::make_unique<MLP::Layer>(layer) 
		                         + std::make_unique<MLP::Layer>(layer) 
		                         + std::make_unique<MLP::Layer>(MLP::Layer::RandomLayer(20, 5));
	
	std::unique_ptr<Model> model2 = model->clone() + std::make_unique<MLP::Layer>(MLP::Layer::RandomLayer(5, 1));


	model->setInput(DeviceMatrix::Random(20, 1, { -10.0f, 1.0f }));

	printf("%zu", model->depth());

    return 0;
}

