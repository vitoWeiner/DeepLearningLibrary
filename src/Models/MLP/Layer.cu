#include "../../../include/Models/MLP/Layer.cuh"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <utility> // for std::pair
#include <cstddef> // for size_t
#include <memory>


namespace dl {

	namespace MLP {

        Layer::Layer() {}
        
        Layer::Layer(const DeviceMatrix& Weights, const DeviceMatrix& Biases) {
        
        	if (Weights.rows() != Biases.rows())
        		throw std::runtime_error("Weights and Biases must have the same number of rows");
        
        	this->weights = Weights;
        	this->biases = Biases;
        	//this->input_size = Weights.cols();
        	//this->output_size = Weights.rows();
        
        }
        
        
        
        Layer::Layer(DeviceMatrix&& Weights, DeviceMatrix&& Biases) {
        
        	if (Weights.rows() != Biases.rows())
        		throw std::runtime_error("Weights and Biases must have the same number of rows");
        
        	this->weights = std::move(Weights);
        	this->biases = std::move(Biases);
        	//this->input_size = this->weights.cols();
        	//this->output_size = this->weights.rows();
        }
        
        /*Layer::Layer(const Layer& layer) :
        	weights(layer.weights),
        	biases(layer.biases),
        	input(layer.input)
            {}*/
        	//input_size(layer.input_size),
        	//output_size(layer.output_size) 
        
        
        
        /*Layer& Layer::operator=(const Layer& layer) {
        	if (this == &layer) {
        		return *this;
        	}
        	this->weights = layer.weights;
        	this->biases = layer.biases;
        	//this->input_size = layer.input_size;
        	//this->output_size = layer.output_size;
        	return *this;
        }*/
        
        
        void Layer::setInput(const DeviceMatrix& input_matrix) {
        
        	if (input_matrix.rows() != this->inputSize())
        		throw std::runtime_error("Input matrix must have the same number of rows as the layer's input size");
        
        	this->input = input_matrix;
        }
        
        void Layer::setInput(DeviceMatrix&& input_matrix) {
        	if (input_matrix.rows() != this->inputSize())
        		throw std::runtime_error("Input matrix must have the same number of rows as the layer's input size");
        
        	this->input = std::move(input_matrix);
        }
        
        
        /*Layer::Layer(Layer&& layer) noexcept :
        	weights(std::move(layer.weights)),
        	biases(std::move(layer.biases)),
        	input(std::move(layer.input))
        {
        	//layer.input_size = 0;
        	//layer.output_size = 0;
        }*/
        	//input_size(layer.input_size),
        	//output_size(layer.output_size) 
        
        
       /* Layer& Layer::operator=(Layer&& layer) noexcept {
        	if (this == &layer) {
        		return *this;
        	}
        	this->weights = std::move(layer.weights);
        	this->biases = std::move(layer.biases);
        	this->input = std::move(layer.input);
        	//this->input_size = layer.input_size;
        	//this->output_size = layer.output_size;
        	//layer.input_size = 0;
        	//layer.output_size = 0;
        	return *this;
        }*/
        
        
        DeviceMatrix Layer::forward() {
        
        	if (this->input.totalSize() == 0)
        		throw std::runtime_error("Input matrix is empty");
        
        	
        	// W*x ++ biases;
        
        	DeviceMatrix output = DeviceMatrix::matMul(this->weights, this->input);
        	output = DeviceMatrix::broadcastAdd(output, this->biases);
        
        	return output;
        
        }
        
        DeviceMatrix Layer::backpropagate(DeviceMatrix nablaC) {  // backpropagation only, no training step
        	if (nablaC.rows() != this->outputSize())
        		throw std::runtime_error("Gradient output must have the same number of rows as the layer's output size");
        
        	return DeviceMatrix::matMul(DeviceMatrix::matTranspose(this->weights), nablaC);
        }
        
        DeviceMatrix Layer::updateParamsAndBackpropagate(DeviceMatrix gradient_output, float learning_rate) {  // backpropagation + training step
        
        	if (gradient_output.rows() != this->outputSize())
        		throw std::runtime_error("Gradient output must have the same number of rows as the layer's output size");
        
        	// gradients calculation
        	DeviceMatrix gradient_weights = DeviceMatrix::matMul(gradient_output, DeviceMatrix::matTranspose(this->input));
        	DeviceMatrix gradient_biases = DeviceMatrix::matColReduce(gradient_output);
        	DeviceMatrix gradient_input = this->backpropagate(gradient_output);
        
        	// scaling gradients
        	gradient_weights = DeviceMatrix::matScale(gradient_weights, learning_rate);
        	gradient_biases = DeviceMatrix::matScale(gradient_biases, learning_rate);
        
        	// training step
        	weights = DeviceMatrix::matSub(weights, gradient_weights);
        	biases = DeviceMatrix::matSub(biases, gradient_biases);
        
        	return gradient_input;
        }
        
        
        void Layer::clean() noexcept {
        	this->weights.clean();
        	this->biases.clean();
        	this->input.clean();
        	//this->input_size = 0;
        	//this->output_size = 0;
        }
        

        Layer Layer::RandomLayer(size_t input_size, size_t output_size, std::pair<float, float> range) {
        
        	if (input_size == 0 || output_size == 0)
        		throw std::runtime_error("Input and output sizes must be greater than zero");
        
        	DeviceMatrix Weights = DeviceMatrix::Random(output_size, input_size, range);
        	DeviceMatrix Biases = DeviceMatrix::Random(output_size, 1, range);
        
        	return Layer(std::move(Weights), std::move(Biases));
        }

        void Layer::print(const char* header) const {
            
            Matrix weights_host = this->weights.downloadToHost();
            Matrix biases_host = this->biases.downloadToHost();
            
            printf("%s\n__________\n", header);
			printf("input size: %zu\n", this->inputSize());
			printf("output size: %zu\n", this->outputSize());
			printf("parameter count: %zu\n", this->parameterCount());
            printf("\n__________\n");
            
            printf("Weights:\n\n");

            weights_host.print(10, 10);
            

            printf("\n__________\nBiases:\n\n");
            biases_host.print(10, 10);
			printf("\n__________\n\n");
		}

        std::shared_ptr<LearningUnit> Layer::clone() const {
            return std::make_shared<Layer>(*this);
        }
        
	}; // namespace MLP      
};  // namespace dl