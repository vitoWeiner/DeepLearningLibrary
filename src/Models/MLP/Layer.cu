#include "../../../include/Models/MLP/Layer.cuh"
#include <hip/hip_runtime.h>
#include <stdexcept>


MLP::Layer::Layer() : input_size(0), output_size(0) {}

MLP::Layer::Layer(const DeviceMatrix& Weights, const DeviceMatrix& Biases) {

	if (Weights.rows() != Biases.rows())
		throw std::runtime_error("Weights and Biases must have the same number of rows");

	this->weights = Weights;
	this->biases = Biases;
	this->input_size = Weights.cols();
	this->output_size = Weights.rows();

}



MLP::Layer::Layer(DeviceMatrix&& Weights, DeviceMatrix&& Biases) {

	if (Weights.rows() != Biases.rows())
		throw std::runtime_error("Weights and Biases must have the same number of rows");

	this->weights = std::move(Weights);
	this->biases = std::move(Biases);
	this->input_size = this->weights.cols();
	this->output_size = this->weights.rows();
}

MLP::Layer::Layer(const Layer& layer) :
	weights(layer.weights),
	biases(layer.biases),
	input(layer.input),
	input_size(layer.input_size),
	output_size(layer.output_size) {}



MLP::Layer& MLP::Layer::operator=(const Layer& layer) {
	if (this == &layer) {
		return *this;
	}
	this->weights = layer.weights;
	this->biases = layer.biases;
	this->input_size = layer.input_size;
	this->output_size = layer.output_size;
	return *this;
}


void MLP::Layer::setInput(const DeviceMatrix& input_matrix) {

	if (input_matrix.rows() != this->input_size)
		throw std::runtime_error("Input matrix must have the same number of rows as the layer's input size");

	this->input = input_matrix;
}

void MLP::Layer::setInput(DeviceMatrix&& input_matrix) {
	if (input_matrix.rows() != this->input_size)
		throw std::runtime_error("Input matrix must have the same number of rows as the layer's input size");

	this->input = std::move(input_matrix);
}


MLP::Layer::Layer(Layer&& layer) noexcept :
	weights(std::move(layer.weights)),
	biases(std::move(layer.biases)),
	input(std::move(layer.input)),
	input_size(layer.input_size),
	output_size(layer.output_size) {
	layer.input_size = 0;
	layer.output_size = 0;
}

MLP::Layer& MLP::Layer::operator=(Layer&& layer) noexcept {
	if (this == &layer) {
		return *this;
	}
	this->weights = std::move(layer.weights);
	this->biases = std::move(layer.biases);
	this->input = std::move(layer.input);
	this->input_size = layer.input_size;
	this->output_size = layer.output_size;
	layer.input_size = 0;
	layer.output_size = 0;
	return *this;
}


DeviceMatrix MLP::Layer::forward() {

	if (this->input.totalSize() == 0)
		throw std::runtime_error("Input matrix is empty");

	
	// W*x ++ biases;

	DeviceMatrix output = DeviceMatrix::matMul(this->weights, this->input);
	output = DeviceMatrix::broadcastAdd(output, this->biases);

	return output;

}

DeviceMatrix MLP::Layer::backpropagate(const DeviceMatrix& nablaC) {  // backpropagation only, no training step
	if (nablaC.rows() != this->output_size)
		throw std::runtime_error("Gradient output must have the same number of rows as the layer's output size");

	return DeviceMatrix::matMul(DeviceMatrix::matTranspose(this->weights), nablaC);
}

DeviceMatrix MLP::Layer::train(const DeviceMatrix& gradient_output, float learning_rate) {  // backpropagation + training step

	if (gradient_output.rows() != this->output_size)
		throw std::runtime_error("Gradient output must have the same number of rows as the layer's output size");

	// gradients calculation
	DeviceMatrix gradient_weights = DeviceMatrix::matMul(gradient_output, DeviceMatrix::matTranspose(input));
	DeviceMatrix gradient_biases = DeviceMatrix::matColReduce(gradient_output);
	DeviceMatrix gradient_input = this->backpropagate(gradient_output);

	// scaling gradients
	gradient_weights = DeviceMatrix::matScale(gradient_weights, learning_rate);
	gradient_biases = DeviceMatrix::matScale(gradient_biases, learning_rate);

	// training step
	weights = DeviceMatrix::matSub(weights, gradient_weights);
	biases = DeviceMatrix::matSub(biases, gradient_biases);

	return gradient_input;
}


void MLP::Layer::clean() noexcept {
	this->weights.clean();
	this->biases.clean();
	this->input.clean();
	this->input_size = 0;
	this->output_size = 0;
}


MLP::Layer MLP::Layer::RandomLayer(size_t input_size, size_t output_size, std::pair<float, float> range) {

	if (input_size == 0 || output_size == 0)
		throw std::runtime_error("Input and output sizes must be greater than zero");

	DeviceMatrix Weights = DeviceMatrix::Random(output_size, input_size, range);
	DeviceMatrix Biases = DeviceMatrix::Random(output_size, 1, range);

	return Layer(Weights, Biases);
}