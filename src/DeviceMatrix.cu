#include "hip/hip_runtime.h"
// file: DeviceMatrix.cu, ovdje se nalaze metode za rad s DeviceMatrix preko hosta ali ne i preko devicea (nema kernela)


#include "../include/DeviceMatrix.cuh"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <algorithm>  
#include <initializer_list>

namespace dl {


    long long DeviceMatrix::instances = 0;
    long long DeviceMatrix::ID = 0;

    std::unordered_set<long long> DeviceMatrix::ids;


    DeviceMatrix::DeviceMatrix() :
        device_matrix(nullptr), rows_count(0), cols_count(0), total_size(0), id(-10) {
        //DeviceMatrix::increment();
    }

    DeviceMatrix::DeviceMatrix(const DeviceMatrix& other) :
        device_matrix(nullptr), rows_count(other.rows_count), cols_count(other.cols_count), total_size(other.total_size), id(-20) {

        if (other.device_matrix == nullptr) {
            return;
        }

        // else if (other.device_matrix is not null )

        hipError_t cuda_malloc_error = hipMalloc(&device_matrix, this->total_size * sizeof(float));

        if (cuda_malloc_error != hipSuccess) {
            throw std::runtime_error("DeviceMatrix copy constructor error:\n hipMalloc failed: " + std::string(hipGetErrorString(cuda_malloc_error)));

        }

        hipError_t cuda_memcpy_error = hipMemcpy(device_matrix, other.device_matrix, this->total_size * sizeof(float), hipMemcpyDeviceToDevice);

        if (cuda_memcpy_error != hipSuccess) {
            hipFree(this->device_matrix);
            throw std::runtime_error("DeviceMatrix copy constructor error:\n hipMemcpy failed: " + std::string(hipGetErrorString(cuda_memcpy_error)));
        }

        DeviceMatrix::increment(this);
    }

    DeviceMatrix& DeviceMatrix::operator=(const DeviceMatrix& other)
    {
        if (this == &other) {
            return *this;
        }

        if (this->device_matrix) {
            hipError_t err = hipFree(this->device_matrix);
            if (err != hipSuccess) {
                //throw std::runtime_error("DeviceMatrix assignment error: hipFree failed: " + std::string(hipGetErrorString(err)));
                printf("[DeviceMatrix] Warning: hipFree failed in assignment: %s\n", hipGetErrorString(err));
            }
            this->device_matrix = nullptr;
            DeviceMatrix::decrement(this->id);
        }

        this->rows_count = other.rows_count;
        this->cols_count = other.cols_count;
        this->total_size = other.total_size;

        if (other.device_matrix == nullptr) {
            this->device_matrix = nullptr;
            return *this;
        }

        // else if ( other.device_matrix is not null )

        hipError_t cuda_malloc_error = hipMalloc(&this->device_matrix, this->total_size * sizeof(float));

        if (cuda_malloc_error != hipSuccess) {
            throw std::runtime_error("DeviceMatrix assignment error:\n hipMalloc failed: " + std::string(hipGetErrorString(cuda_malloc_error)));
        }

        hipError_t cuda_memcpy_error = hipMemcpy(this->device_matrix, other.device_matrix, this->total_size * sizeof(float), hipMemcpyDeviceToDevice);

        if (cuda_memcpy_error != hipSuccess) {
            hipFree(this->device_matrix);
            throw std::runtime_error("DeviceMatrix assignment error:\n hipMemcpy failed: " + std::string(hipGetErrorString(cuda_memcpy_error)));
        }

        DeviceMatrix::increment(this);

        return *this;
    }

    DeviceMatrix::DeviceMatrix(const Matrix& mat) :
        device_matrix(nullptr), rows_count(mat.rows()), cols_count(mat.cols()), total_size(mat.totalSize()) {

        const float* temp_arr = mat.borrowData();

        if (temp_arr == nullptr) {
            return;
        }

        // else if (temp_arr is not null )

        hipError_t cuda_malloc_error = hipMalloc(&device_matrix, this->total_size * sizeof(float));

        if (cuda_malloc_error != hipSuccess) {
            throw std::runtime_error("error from CUDA_Matrix constructor-0:\n hipMalloc failed:\n " + std::string(hipGetErrorString(cuda_malloc_error)));
        }

        hipError_t cuda_memcpy_error = hipMemcpy(device_matrix, temp_arr, this->total_size * sizeof(float), hipMemcpyHostToDevice);

        if (cuda_memcpy_error != hipSuccess) {

            hipFree(device_matrix);
            throw std::runtime_error("error in CUDA_Matrix.uploadFromMatrix(Matrix M): problem:\n hipMemcpy failed:\n" + std::string(hipGetErrorString(cuda_memcpy_error)));
        }

        DeviceMatrix::increment(this);
    }

    DeviceMatrix::DeviceMatrix(std::initializer_list<float> args, size_t rows, size_t cols) :
        device_matrix(nullptr), rows_count(rows), cols_count(cols), total_size(rows* cols)
    {

        if (args.size() != this->total_size) {
            throw std::invalid_argument("DeviceMatrix constructor error: initializer list size does not match dimensions.");
        }

        if (this->total_size == 0) {
            return;
        }

        // else if (this->total_size is not 0)

        float* temp_arr = new float[this->total_size];

        std::copy(args.begin(), args.end(), temp_arr);

        hipError_t cuda_malloc_error = hipMalloc(&this->device_matrix, this->total_size * sizeof(float));

        if (cuda_malloc_error != hipSuccess) {
            delete[] temp_arr;
            throw std::runtime_error("DeviceMatrix constructor error:\n hipMalloc failed: " + std::string(hipGetErrorString(cuda_malloc_error)));
        }

        hipError_t cuda_memcpy_error = hipMemcpy(this->device_matrix, temp_arr, this->total_size * sizeof(float), hipMemcpyHostToDevice);

        delete[] temp_arr;

        if (cuda_memcpy_error != hipSuccess) {
            hipFree(this->device_matrix);
            throw std::runtime_error("DeviceMatrix constructor error:\n hipMemcpy failed: " + std::string(hipGetErrorString(cuda_memcpy_error)));
        }

        DeviceMatrix::increment(this);
    }



    DeviceMatrix::DeviceMatrix(size_t rows, size_t cols) :
        device_matrix(nullptr), rows_count(rows), cols_count(cols), total_size(rows* cols)
    {

        if (this->total_size == 0) {
            return;
        }

        // else if (this->total_size is not 0)

        hipError_t cuda_malloc_error = hipMalloc(&this->device_matrix, this->total_size * sizeof(float));

        if (cuda_malloc_error != hipSuccess) {
            throw std::runtime_error("DeviceMatrix constructor error:\n hipMalloc failed: " + std::string(hipGetErrorString(cuda_malloc_error)));
        }

        hipError_t cuda_memset_error = hipMemset(this->device_matrix, 0, this->total_size * sizeof(float));

        if (cuda_memset_error != hipSuccess) {
            hipFree(this->device_matrix);
            throw std::runtime_error("DeviceMatrix constructor error:\n hipMemset failed: " + std::string(hipGetErrorString(cuda_memset_error)));
        }

        DeviceMatrix::increment(this);
    }

    //#include <iostream>

    DeviceMatrix::DeviceMatrix(DeviceMatrix&& other) noexcept
        : device_matrix(other.device_matrix),
        rows_count(other.rows_count),
        cols_count(other.cols_count),
        total_size(other.total_size),
        id(other.id)  // Move the id as well
    {

        //std::cout << "[Move ctor]" << std::endl;
        other.device_matrix = nullptr;
        other.rows_count = 0;
        other.cols_count = 0;
        other.total_size = 0;
    }



    DeviceMatrix& DeviceMatrix::operator=(DeviceMatrix&& other) noexcept { // error handlinga nemna nego log

        //std::cout << "[Move assignment operator]" << std::endl;

        if (this == &other) {
            return *this;
        }

        if (this->device_matrix) {
            hipError_t err = hipFree(this->device_matrix);
            DeviceMatrix::decrement(this->id);

            if (err != hipSuccess) {
                //std::cerr << "[DeviceMatrix] Warning: hipFree failed in move assignment: "
                          // << hipGetErrorString(err) << std::endl;
            }
        }

        this->device_matrix = other.device_matrix;
        this->rows_count = other.rows_count;
        this->cols_count = other.cols_count;
        this->total_size = other.total_size;
        this->id = other.id;  // Move the id as well

        other.device_matrix = nullptr;
        other.rows_count = 0;
        other.cols_count = 0;
        other.total_size = 0;

        return *this;
    }




    size_t DeviceMatrix::rows() const noexcept {
        return this->rows_count;
    }

    size_t DeviceMatrix::cols() const noexcept {
        return this->cols_count;
    }

    size_t DeviceMatrix::totalSize() const noexcept {
        return this->total_size;
    }

    const float* DeviceMatrix::borrowData() const noexcept {
        return this->device_matrix;
    }


    Matrix DeviceMatrix::downloadToHost() const {  // moram dodat error handling

        if (this->device_matrix == nullptr) {
            return Matrix(0, 0); // Return an empty Matrix if device_matrix is null 
        }

        float* host_data = new float[total_size];

        hipError_t err = hipMemcpy(host_data, device_matrix, total_size * sizeof(float), hipMemcpyDeviceToHost);

        if (err != hipSuccess) {
            delete[] host_data;
            throw std::runtime_error("DeviceMatrix constructor error:\n hipMemcpy failed: " + std::string(hipGetErrorString(err)));
        }

        Matrix result(host_data, rows_count, cols_count);

        delete[] host_data;

        return result;
    }

    DeviceMatrix DeviceMatrix::matConcatCols(const DeviceMatrix& A, const DeviceMatrix& B) {

        if (A.rows() != B.rows())
            throw std::runtime_error("in function matConcatCols : runtime error : A.rows != B.rows");

        DeviceMatrix result(A.rows(), A.cols() + B.cols());


        // bytes_in_row_* = every row of matrix * has same number of elements -> same number of bytes, this is how much
        size_t bytes_in_row_A = A.cols() * sizeof(float);
        size_t bytes_in_row_B = B.cols() * sizeof(float);
        size_t bytes_in_row_result = result.cols() * sizeof(float);

        for (size_t row = 0; row < A.rows(); ++row) {

            hipError_t err = hipMemcpy(result.device_matrix + row * result.cols(), A.device_matrix + row * A.cols(), bytes_in_row_A, hipMemcpyDeviceToDevice);

            if (err != hipSuccess) {
                throw std::runtime_error("in function DeviceMatrix::matConcatCols hipMemcpy failed : " + std::string(hipGetErrorString(err)));
            }


            err = hipMemcpy(result.device_matrix + row * result.cols() + A.cols(), B.device_matrix + row * B.cols(), bytes_in_row_B, hipMemcpyDeviceToDevice);

            if (err != hipSuccess) {
                throw std::runtime_error("in function DeviceMatrix::matConcatCols hipMemcpy failed : " + std::string(hipGetErrorString(err)));
            }
        }


        return result;
    }

    DeviceMatrix DeviceMatrix::matConcatRows(const DeviceMatrix& A, const DeviceMatrix& B) {
        if (A.cols() != B.cols())
            throw std::runtime_error("in function matConcatRows : runtime error : A.cols() != B.cols()");

        DeviceMatrix result(A.rows() + B.rows(), A.cols());

        hipError_t err = hipMemcpy(result.device_matrix, A.device_matrix, A.totalSize() * sizeof(float), hipMemcpyDeviceToDevice);

        if (err != hipSuccess) {
            throw std::runtime_error("in function DeviceMatrix::matConcatRows : hipMemcpy failed : " + std::string(hipGetErrorString(err)));
        }

        err = hipMemcpy(result.device_matrix + A.totalSize(), B.device_matrix, B.totalSize() * sizeof(float), hipMemcpyDeviceToDevice);

        if (err != hipSuccess) {
            throw std::runtime_error("in function DeviceMatrix::matConcatRows : hipMemcpy failed : " + std::string(hipGetErrorString(err)));
        }

        return result;
    }

void DeviceMatrix::downloadToHost(float* buffer) const {  // moram dodat error handling

    if (this->device_matrix == nullptr) {
        return; // Return an empty Matrix if device_matrix is null 
    }



    hipError_t err = hipMemcpy(buffer, device_matrix, total_size * sizeof(float), hipMemcpyDeviceToHost);

    if (err != hipSuccess) {

        throw std::runtime_error("DeviceMatrix constructor error:\n hipMemcpy failed: " + std::string(hipGetErrorString(err)));
    }

    return;
}


void DeviceMatrix::clean() noexcept {
    if (this->device_matrix != nullptr) {

        hipError_t err = hipFree(this->device_matrix);
        if (err != hipSuccess) {
            //std::cerr << "[DeviceMatrix] Warning: hipFree failed in clean: " << hipGetErrorString(err) << std::endl;
        }

        this->device_matrix = nullptr;
        this->rows_count = 0;
        this->cols_count = 0;
        this->total_size = 0;
	    DeviceMatrix::decrement(this->id);
    }

}

DeviceMatrix::~DeviceMatrix() {
    if (device_matrix != nullptr) {
        hipFree(device_matrix);
		DeviceMatrix::decrement(this->id);
    }
}

};