// file: DeviceMatrix.cu, ovdje se nalaze metode za rad s DeviceMatrix preko hosta ali ne i preko devicea (nema kernela)


#include "../include/DeviceMatrix.cuh"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <algorithm>  
#include <initializer_list>

namespace dl {


    DeviceMatrix::DeviceMatrix() :
        device_matrix(nullptr), rows_count(0), cols_count(0), total_size(0) {
      
    }

    DeviceMatrix::DeviceMatrix(const DeviceMatrix& other) :
        device_matrix(nullptr), rows_count(other.rows_count), cols_count(other.cols_count), total_size(other.total_size) {

        if (other.device_matrix == nullptr) {
            return;
        }

       

        hipError_t cuda_malloc_error = hipMalloc(&device_matrix, this->total_size * sizeof(float));

        if (cuda_malloc_error != hipSuccess) {
            throw std::runtime_error("DeviceMatrix copy constructor error:\n hipMalloc failed: " + std::string(hipGetErrorString(cuda_malloc_error)));

        }

        hipError_t cuda_memcpy_error = hipMemcpy(device_matrix, other.device_matrix, this->total_size * sizeof(float), hipMemcpyDeviceToDevice);

        if (cuda_memcpy_error != hipSuccess) {
            hipFree(this->device_matrix);
            throw std::runtime_error("DeviceMatrix copy constructor error:\n hipMemcpy failed: " + std::string(hipGetErrorString(cuda_memcpy_error)));
        }

     
    }

    DeviceMatrix& DeviceMatrix::operator=(const DeviceMatrix& other)
    {
        if (this == &other) {
            return *this;
        }

        if (this->device_matrix) {
            hipError_t err = hipFree(this->device_matrix);
            if (err != hipSuccess) {
                
                printf("[DeviceMatrix] Warning: hipFree failed in assignment: %s\n", hipGetErrorString(err));
            }
            this->device_matrix = nullptr;
           
        }

        this->rows_count = other.rows_count;
        this->cols_count = other.cols_count;
        this->total_size = other.total_size;

        if (other.device_matrix == nullptr) {
            this->device_matrix = nullptr;
            return *this;
        }

       

        hipError_t cuda_malloc_error = hipMalloc(&this->device_matrix, this->total_size * sizeof(float));

        if (cuda_malloc_error != hipSuccess) {
            throw std::runtime_error("DeviceMatrix assignment error:\n hipMalloc failed: " + std::string(hipGetErrorString(cuda_malloc_error)));
        }

        hipError_t cuda_memcpy_error = hipMemcpy(this->device_matrix, other.device_matrix, this->total_size * sizeof(float), hipMemcpyDeviceToDevice);

        if (cuda_memcpy_error != hipSuccess) {
            hipFree(this->device_matrix);
            throw std::runtime_error("DeviceMatrix assignment error:\n hipMemcpy failed: " + std::string(hipGetErrorString(cuda_memcpy_error)));
        }

       

        return *this;
    }

    DeviceMatrix::DeviceMatrix(const Matrix& mat) :
        device_matrix(nullptr), rows_count(mat.rows()), cols_count(mat.cols()), total_size(mat.totalSize()) {

        const float* temp_arr = mat.borrowData();

        if (temp_arr == nullptr) {
            return;
        }

       

        hipError_t cuda_malloc_error = hipMalloc(&device_matrix, this->total_size * sizeof(float));

        if (cuda_malloc_error != hipSuccess) {
            throw std::runtime_error("error from CUDA_Matrix constructor-0:\n hipMalloc failed:\n " + std::string(hipGetErrorString(cuda_malloc_error)));
        }

        hipError_t cuda_memcpy_error = hipMemcpy(device_matrix, temp_arr, this->total_size * sizeof(float), hipMemcpyHostToDevice);

        if (cuda_memcpy_error != hipSuccess) {

            hipFree(device_matrix);
            throw std::runtime_error("error in CUDA_Matrix.uploadFromMatrix(Matrix M): problem:\n hipMemcpy failed:\n" + std::string(hipGetErrorString(cuda_memcpy_error)));
        }

      
    }

    DeviceMatrix::DeviceMatrix(std::initializer_list<float> args, size_t rows, size_t cols) :
        device_matrix(nullptr), rows_count(rows), cols_count(cols), total_size(rows* cols)
    {

        if (args.size() != this->total_size) {
            throw std::invalid_argument("DeviceMatrix constructor error: initializer list size does not match dimensions.");
        }

        if (this->total_size == 0) {
            return;
        }

     

        float* temp_arr = new float[this->total_size];

        std::copy(args.begin(), args.end(), temp_arr);

        hipError_t cuda_malloc_error = hipMalloc(&this->device_matrix, this->total_size * sizeof(float));

        if (cuda_malloc_error != hipSuccess) {
            delete[] temp_arr;
            throw std::runtime_error("DeviceMatrix constructor error:\n hipMalloc failed: " + std::string(hipGetErrorString(cuda_malloc_error)));
        }

        hipError_t cuda_memcpy_error = hipMemcpy(this->device_matrix, temp_arr, this->total_size * sizeof(float), hipMemcpyHostToDevice);

        delete[] temp_arr;

        if (cuda_memcpy_error != hipSuccess) {
            hipFree(this->device_matrix);
            throw std::runtime_error("DeviceMatrix constructor error:\n hipMemcpy failed: " + std::string(hipGetErrorString(cuda_memcpy_error)));
        }

       
    }



    DeviceMatrix::DeviceMatrix(size_t rows, size_t cols) :
        device_matrix(nullptr), rows_count(rows), cols_count(cols), total_size(rows* cols)
    {

        if (this->total_size == 0) {
            return;
        }

       

        hipError_t cuda_malloc_error = hipMalloc(&this->device_matrix, this->total_size * sizeof(float));

        if (cuda_malloc_error != hipSuccess) {
            throw std::runtime_error("DeviceMatrix constructor error:\n hipMalloc failed: " + std::string(hipGetErrorString(cuda_malloc_error)));
        }

        hipError_t cuda_memset_error = hipMemset(this->device_matrix, 0, this->total_size * sizeof(float));

        if (cuda_memset_error != hipSuccess) {
            hipFree(this->device_matrix);
            throw std::runtime_error("DeviceMatrix constructor error:\n hipMemset failed: " + std::string(hipGetErrorString(cuda_memset_error)));
        }

       
    }

    

    DeviceMatrix::DeviceMatrix(DeviceMatrix&& other) noexcept
        : device_matrix(other.device_matrix),
        rows_count(other.rows_count),
        cols_count(other.cols_count),
        total_size(other.total_size)  
    {

      
        other.device_matrix = nullptr;
        other.rows_count = 0;
        other.cols_count = 0;
        other.total_size = 0;
    }



    DeviceMatrix& DeviceMatrix::operator=(DeviceMatrix&& other) noexcept { 

       

        if (this == &other) {
            return *this;
        }

        if (this->device_matrix) {
            hipError_t err = hipFree(this->device_matrix);
           

            if (err != hipSuccess) {
                
            }
        }

        this->device_matrix = other.device_matrix;
        this->rows_count = other.rows_count;
        this->cols_count = other.cols_count;
        this->total_size = other.total_size;
       

        other.device_matrix = nullptr;
        other.rows_count = 0;
        other.cols_count = 0;
        other.total_size = 0;

        return *this;
    }




    size_t DeviceMatrix::rows() const noexcept {
        return this->rows_count;
    }

    size_t DeviceMatrix::cols() const noexcept {
        return this->cols_count;
    }

    size_t DeviceMatrix::totalSize() const noexcept {
        return this->total_size;
    }

    const float* DeviceMatrix::borrowData() const noexcept {
        return this->device_matrix;
    }


    Matrix DeviceMatrix::downloadToHost() const { 

        if (this->device_matrix == nullptr) {
            return Matrix(0, 0); 
        }

        float* host_data = new float[total_size];

        hipError_t err = hipMemcpy(host_data, device_matrix, total_size * sizeof(float), hipMemcpyDeviceToHost);

        if (err != hipSuccess) {
            delete[] host_data;
            throw std::runtime_error("DeviceMatrix constructor error:\n hipMemcpy failed: " + std::string(hipGetErrorString(err)));
        }

        Matrix result(host_data, rows_count, cols_count);

        delete[] host_data;

        return result;
    }

    DeviceMatrix DeviceMatrix::matConcatCols(const DeviceMatrix& A, const DeviceMatrix& B) {

        if (A.rows() != B.rows())
            throw std::runtime_error("in function matConcatCols : runtime error : A.rows != B.rows");

        DeviceMatrix result(A.rows(), A.cols() + B.cols());


        // bytes_in_row_* = every row of matrix * has same number of elements -> same number of bytes, this is how much
        size_t bytes_in_row_A = A.cols() * sizeof(float);
        size_t bytes_in_row_B = B.cols() * sizeof(float);
        size_t bytes_in_row_result = result.cols() * sizeof(float);

        for (size_t row = 0; row < A.rows(); ++row) {

            hipError_t err = hipMemcpy(result.device_matrix + row * result.cols(), A.device_matrix + row * A.cols(), bytes_in_row_A, hipMemcpyDeviceToDevice);

            if (err != hipSuccess) {
                throw std::runtime_error("in function DeviceMatrix::matConcatCols hipMemcpy failed : " + std::string(hipGetErrorString(err)));
            }


            err = hipMemcpy(result.device_matrix + row * result.cols() + A.cols(), B.device_matrix + row * B.cols(), bytes_in_row_B, hipMemcpyDeviceToDevice);

            if (err != hipSuccess) {
                throw std::runtime_error("in function DeviceMatrix::matConcatCols hipMemcpy failed : " + std::string(hipGetErrorString(err)));
            }
        }


        return result;
    }

    DeviceMatrix DeviceMatrix::matConcatRows(const DeviceMatrix& A, const DeviceMatrix& B) {
        if (A.cols() != B.cols())
            throw std::runtime_error("in function matConcatRows : runtime error : A.cols() != B.cols()");

        DeviceMatrix result(A.rows() + B.rows(), A.cols());

        hipError_t err = hipMemcpy(result.device_matrix, A.device_matrix, A.totalSize() * sizeof(float), hipMemcpyDeviceToDevice);

        if (err != hipSuccess) {
            throw std::runtime_error("in function DeviceMatrix::matConcatRows : hipMemcpy failed : " + std::string(hipGetErrorString(err)));
        }

        err = hipMemcpy(result.device_matrix + A.totalSize(), B.device_matrix, B.totalSize() * sizeof(float), hipMemcpyDeviceToDevice);

        if (err != hipSuccess) {
            throw std::runtime_error("in function DeviceMatrix::matConcatRows : hipMemcpy failed : " + std::string(hipGetErrorString(err)));
        }

        return result;
    }

void DeviceMatrix::downloadToHost(float* buffer) const {  // moram dodat error handling

    if (this->device_matrix == nullptr) {
        return;
    }



    hipError_t err = hipMemcpy(buffer, device_matrix, total_size * sizeof(float), hipMemcpyDeviceToHost);

    if (err != hipSuccess) {

        throw std::runtime_error("DeviceMatrix constructor error:\n hipMemcpy failed: " + std::string(hipGetErrorString(err)));
    }

    return;
}


void DeviceMatrix::clean() noexcept {
    if (this->device_matrix != nullptr) {

        hipError_t err = hipFree(this->device_matrix);
        if (err != hipSuccess) {
            //std::cerr << "[DeviceMatrix] Warning: hipFree failed in clean: " << hipGetErrorString(err) << std::endl;
        }

        this->device_matrix = nullptr;
        this->rows_count = 0;
        this->cols_count = 0;
        this->total_size = 0;
	   
    }

}

DeviceMatrix::~DeviceMatrix() {
    if (device_matrix != nullptr) {
        hipFree(device_matrix);

    }
}

};