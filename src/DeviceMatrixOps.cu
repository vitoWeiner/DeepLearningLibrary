#include "hip/hip_runtime.h"
// file: DeviceMatrixOps.cu, ovdje se nalaze metode za rad s DeviceMatrix preko devicea (kerneli)


/*

POZOR:

NEKI KERNELI NISU JOS OPTIMIZIRANI NEGO TRIVIJALNO IMPLEMENTIRANI, SEMANTICKI ISPRAVNI, RADE ISPRAVNO, ALI POTPUNA OPTIMIZACIJA JOS NIJE IZVEDENA.
NPR:
- reduction sum radi ali treba dodati umjesto trivijalnog algoritma, algoritam redukcije putem stabla redukcije, a ne trivijalno zbrajanje svakog retka.
- matmul je dosta optimiziran, radi ispravno, no postoje izvori sporosti poput bank konflikata, to jos treba rjesiti, neki problemi s bank konfliktima i mozda dodati jos koje optimizacije.
- banchmarking i optimiziranje velicine blokova i grida treba isto, trenutno su simbolicni 2x2, u praksi je cesto 16x16, ali treba testirati na uredaju i vidjeti sto je najoptimalnije.

*/


#include "./../include/DeviceMatrix.cuh"

#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

#include <stdexcept>
#include <cstring>
#include <iostream>


#define TILE_WIDTH 16
#define TILE_HEIGHT 16


namespace dl {

__global__ void tiled_matmul(float* res, float* X, float* Y, int rowsX, int colsX, int colsY) {

    //assert(TILE_WIDTH == blockDim.x && TILE_HEIGHT == blockDim.y && TILE_WIDTH == TILE_HEIGHT);

    const int block_idx_row = blockDim.y * blockIdx.y; // indeks na prvi thread bloka (redak)
    const int block_idx_col = blockDim.x * blockIdx.x; // indeks na prvi thread bloka (stupac)

    int tiles_vector_size = (colsX + TILE_WIDTH - 1) / TILE_WIDTH; // koliko tileova sadrzi tile row od X odnosno tile col od Y. to je ceil(colsX / TILE_WIDTH)
    // jer imamo situaciju ceil(a, b) = (a + b - 1) / b; ovo sljedi iz toga sto 1.) ako je a djeljiv s b, a+b-1 nece preci sljedeci cijeli broj te ce rezultat djeljenja biti isti. No ako je ostatak bar 1, preci ce sljedeci cjeli broj i nece biti isti.

    __shared__ float sh_tile_x[TILE_HEIGHT][TILE_WIDTH];  // alociranje shared memorije na L1 cache za tile matrice X  (+ 1 radi bank konflikta)
    __shared__ float sh_tile_y[TILE_HEIGHT][TILE_WIDTH];  // alociranje shared memorije na L1 cache za tile matrice Y (+ 1 radi bank konflikta)

    float sum = 0.0f;

    for (int tile_iterator = 0; tile_iterator < tiles_vector_size; ++tile_iterator) {

        // svaki blok pristupa tile redu matrice X i tile stupcu matrice Y koji su jednaki tile_row i tile_col izlazne matrice (block_idx_row, block_idx_col)
        // da bi se izracunalo kojem tileu unutar redka X i kojem tileu unutar stupca Y u trenutnoj iteraciji se pristupa koristi se tile_iterator

        int X_tile_row = block_idx_row;
        int Y_tile_col = block_idx_col;

        int X_tile_col = tile_iterator * TILE_WIDTH;
        int Y_tile_row = tile_iterator * TILE_HEIGHT;

        // sada znamo row i col koordinate prvih elemenata unutar tileova redka X i stupca Y.

        int X_element_row = X_tile_row + threadIdx.y;
        int X_element_col = X_tile_col + threadIdx.x;
        int Y_element_row = Y_tile_row + threadIdx.y;
        int Y_element_col = Y_tile_col + threadIdx.x;


        if (X_element_row < rowsX && X_element_col < colsX) {
            sh_tile_x[threadIdx.y][threadIdx.x] = X[X_element_row * colsX + X_element_col]; //d_A_ptr[(row)*A_n_cols + (phase * TILE_WIDTH + tx)];
        }
        else {
            sh_tile_x[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (Y_element_row < colsX && Y_element_col < colsY) {
            sh_tile_y[threadIdx.y][threadIdx.x] = Y[Y_element_row * colsY + Y_element_col]; //d_B_ptr[(phase * TILE_WIDTH + ty) * C_n_cols + (col)];
        }
        else {
            sh_tile_y[threadIdx.y][threadIdx.x] = 0.0f;
        }


        __syncthreads();

        // u ovom trenutku su tileovi ucitani u shared memoriju, vrijeme je za matmul tileova.

        for (int element_iterator = 0; element_iterator < TILE_WIDTH; ++element_iterator) {

            sum += sh_tile_x[threadIdx.y][element_iterator] * sh_tile_y[element_iterator][threadIdx.x];

        }

        __syncthreads();

    }

    int row = block_idx_row + threadIdx.y;
    int col = block_idx_col + threadIdx.x;

    if (row < rowsX && col < colsY) {

        res[row * colsY + col] = sum;

    }

    return;

}


DeviceMatrix DeviceMatrix::matMul(const DeviceMatrix& A, const DeviceMatrix& B) {

    if (A.cols() != B.rows()) {
        throw std::runtime_error("Invalid matrix dimensions for multiplication.");
    }

    if (A.totalSize() == 0 || B.totalSize() == 0) {
        throw std::runtime_error("One of the matrices in matMul is empty.");
    }

    size_t rows = A.rows();
    size_t cols = B.cols();
    size_t shared_dim = A.cols();

    DeviceMatrix result(rows, cols);

    dim3 dimBlock(TILE_WIDTH, TILE_HEIGHT);  // blok mora biti dimenzija istih kao i tileovi, dakle TILE_WIDTH x TILE_HEIGHT
    dim3 dimGrid((cols + TILE_WIDTH - 1) / TILE_WIDTH, (rows + TILE_HEIGHT - 1) / TILE_HEIGHT);  // dimenzije grida odreduju broj blokova u sirinu i visinu, a to je broj stupaca i redaka podijeljen s TILE_WIDTH odnosno TILE_HEIGHT. Koristi se ceil(a, b) = (a + b - 1) / b da bi dobili cijeli broj blokova ako nisu djeljivi (bolje visak pa imamo prazne threadove, nego ne pokriti cijeli).

    tiled_matmul << <dimGrid, dimBlock >> > (
        result.device_matrix,
        A.device_matrix,
        B.device_matrix,
        rows,
        shared_dim,
        cols
        );





    hipError_t err = hipGetLastError();

    hipDeviceSynchronize();

    if (err != hipSuccess) {
        throw std::runtime_error("CUDA kernel launch failed: " + std::string(hipGetErrorString(err)));
    }

    return result;

}



__global__ void trivial_matmul_kernel(float* result, float* X, float* Y, int rowsX, int colsX, int colsY) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= rowsX || col >= colsY)
        return;

    float sum = 0.0f;

    for (int el = 0; el < colsX; ++el) {
        sum += X[row * colsX + el] * Y[el * colsY + col];
    }

    result[row * colsY + col] = sum;
}



DeviceMatrix DeviceMatrix::trivialMatMul(const DeviceMatrix& A, const DeviceMatrix& B) {
    if (A.cols() != B.rows()) {
        throw std::runtime_error("Invalid matrix dimensions for multiplication.");
    }

    if (A.totalSize() == 0 || B.totalSize() == 0) {
        throw std::runtime_error("One of the matrices in matMul is empty.");
    }

    size_t rows = A.rows();
    size_t cols = B.cols();
    size_t shared_dim = A.cols();

    DeviceMatrix result(rows, cols);

    dim3 dimBlock(TILE_WIDTH, TILE_HEIGHT);  // blok mora biti dimenzija istih kao i tileovi, dakle TILE_WIDTH x TILE_HEIGHT
    dim3 dimGrid((cols + TILE_WIDTH - 1) / TILE_WIDTH, (rows + TILE_HEIGHT - 1) / TILE_HEIGHT);

    trivial_matmul_kernel << <dimGrid, dimBlock >> > (
        result.device_matrix,
        A.device_matrix,
        B.device_matrix,
        rows,
        shared_dim,
        cols
        );



    hipError_t err = hipGetLastError();

    hipDeviceSynchronize();

    if (err != hipSuccess) {
        throw std::runtime_error("CUDA kernel launch failed: " + std::string(hipGetErrorString(err)));
    }

    return result;
}



__global__ void trivial_uncoalesced_matmul_kernel(float* result, float* X, float* Y, int rowsX, int colsX, int colsY) {
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= rowsX && col >= colsY)
        return;

    float sum = 0.0f;

    for (int el = 0; el < colsX; ++el) {
        sum += X[row * colsX + el] * Y[el * colsY + col];
    }

    result[row * colsY + col] = sum;
}

DeviceMatrix DeviceMatrix::trivialUncoalescedMatMul(const DeviceMatrix& A, const DeviceMatrix& B) {
    if (A.cols() != B.rows()) {
        throw std::runtime_error("Invalid matrix dimensions for multiplication.");
    }

    if (A.totalSize() == 0 || B.totalSize() == 0) {
        throw std::runtime_error("One of the matrices in matMul is empty.");
    }

    size_t rows = A.rows();
    size_t cols = B.cols();
    size_t shared_dim = A.cols();

    DeviceMatrix result(rows, cols);

    dim3 dimBlock(TILE_WIDTH, TILE_HEIGHT);  // blok mora biti dimenzija istih kao i tileovi, dakle TILE_WIDTH x TILE_HEIGHT
    dim3 dimGrid((rows + TILE_WIDTH - 1) / TILE_WIDTH, (cols + TILE_HEIGHT - 1) / TILE_HEIGHT);

    trivial_uncoalesced_matmul_kernel << <dimGrid, dimBlock >> > (
        result.device_matrix,
        A.device_matrix,
        B.device_matrix,
        rows,
        shared_dim,
        cols
        );



    hipError_t err = hipGetLastError();

    hipDeviceSynchronize();

    if (err != hipSuccess) {
        throw std::runtime_error("CUDA kernel launch failed: " + std::string(hipGetErrorString(err)));
    }

    return result;
}




// Zbrajanje matrica

__global__ void matAdd_kernel(float* result, const float* A, const float* B, size_t rows, size_t cols) {

    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= rows || col >= cols)
        return;

    int idx = row * cols + col;

    result[idx] = A[idx] + B[idx];

}

__global__ void matSub_kernel(float* result, const float* A, const float* B, size_t rows, size_t cols) {

    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= rows || col >= cols)
        return;

    int idx = row * cols + col;

    result[idx] = A[idx] - B[idx];
}

DeviceMatrix DeviceMatrix::matAdd(const DeviceMatrix& A, const DeviceMatrix& B) {

    if (A.rows() != B.rows() || A.cols() != B.cols()) {
        throw std::runtime_error("Matrices must have the same dimensions for addition.");
    }

    if (A.totalSize() == 0 || B.totalSize() == 0) {
        throw std::runtime_error("One of the matrices in matAdd is empty.");
    }

    DeviceMatrix result(A.rows(), A.cols());

    dim3 dimBlock(TILE_WIDTH, TILE_HEIGHT);
    dim3 dimGrid((A.cols() + TILE_WIDTH - 1) / TILE_WIDTH, (A.rows() + TILE_HEIGHT - 1) / TILE_HEIGHT);

    size_t size = A.rows() * A.cols();

    matAdd_kernel << <dimGrid, dimBlock >> > (result.device_matrix, A.device_matrix, B.device_matrix, A.rows(), A.cols());

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();

    if (err != hipSuccess) {
        throw std::runtime_error("CUDA kernel launch failed: " + std::string(hipGetErrorString(err)));
    }

    return result;

}

DeviceMatrix DeviceMatrix::matSub(const DeviceMatrix& A, const DeviceMatrix& B) {

    if (A.rows() != B.rows() || A.cols() != B.cols()) {
        throw std::runtime_error("Matrices must have the same dimensions for subtraction.");
    }

    if (A.totalSize() == 0 || B.totalSize() == 0) {
        throw std::runtime_error("One of the matrices in matSub is empty.");
    }

    DeviceMatrix result(A.rows(), A.cols());
    dim3 dimBlock(TILE_WIDTH, TILE_HEIGHT);
    dim3 dimGrid((A.cols() + TILE_WIDTH - 1) / TILE_WIDTH, (A.rows() + TILE_HEIGHT - 1) / TILE_HEIGHT);
    matSub_kernel << <dimGrid, dimBlock >> > (result.device_matrix, A.device_matrix, B.device_matrix, A.rows(), A.cols());

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();

    if (err != hipSuccess) {
        throw std::runtime_error("CUDA kernel launch failed: " + std::string(hipGetErrorString(err)));
    }

    return result;

}

// element-wise mnozenje matrica

__global__ void elementWiseMultiply_kernel(float* result, const float* A, const float* B, size_t rows, size_t cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= rows || col >= cols)
        return;

    int idx = row * cols + col;
    result[idx] = A[idx] * B[idx];

}

DeviceMatrix DeviceMatrix::matElementWiseMul(const DeviceMatrix& A, const DeviceMatrix& B) {
    if (A.rows() != B.rows() || A.cols() != B.cols()) {
        throw std::runtime_error("Matrices must have the same dimensions for element-wise multiplication.");
    }

    if (A.totalSize() == 0 || B.totalSize() == 0) {
        throw std::runtime_error("One of the matrices in matElementWiseMul is empty.");
    }

    DeviceMatrix result(A.rows(), A.cols());

    dim3 dimBlock(TILE_WIDTH, TILE_HEIGHT);

    dim3 dimGrid((A.cols() + TILE_WIDTH - 1) / TILE_WIDTH, (A.rows() + TILE_HEIGHT - 1) / TILE_HEIGHT);

    elementWiseMultiply_kernel << <dimGrid, dimBlock >> > (result.device_matrix, A.device_matrix, B.device_matrix, A.rows(), A.cols());

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();

    if (err != hipSuccess) {
        throw std::runtime_error("CUDA kernel launch failed: " + std::string(hipGetErrorString(err)));
    }

    return result;
}

// Transponiranje matrice




__global__ void tiled_transpose(float* res, float* X, int rowsX, int colsX) {

    __shared__ float sh_tile[TILE_HEIGHT][TILE_WIDTH + 1]; 

    int row = blockIdx.y * blockDim.y + threadIdx.y;  
    int col = blockIdx.x * blockDim.x + threadIdx.x;  


    if (row < rowsX && col < colsX) {
        sh_tile[threadIdx.y][threadIdx.x] = X[row * colsX + col];
    }

    __syncthreads();


    int transposed_row = blockIdx.x * blockDim.x + threadIdx.y;
    int transposed_col = blockIdx.y * blockDim.y + threadIdx.x;


    if (transposed_row < colsX && transposed_col < rowsX) {
        res[transposed_row * rowsX + transposed_col] = sh_tile[threadIdx.x][threadIdx.y];
    }
}

DeviceMatrix DeviceMatrix::matTranspose(const DeviceMatrix& A) {

    if (A.totalSize() == 0) {
        return A;
    }

    DeviceMatrix output(A.cols(), A.rows());
    dim3 dimBlock(TILE_WIDTH, TILE_HEIGHT);
    dim3 dimGrid((A.cols() + TILE_WIDTH - 1) / TILE_WIDTH, (A.rows() + TILE_HEIGHT - 1) / TILE_HEIGHT);

    tiled_transpose << <dimGrid, dimBlock >> > (
        output.device_matrix,
        A.device_matrix,
        A.rows(),
        A.cols()
        );

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();

    if (err != hipSuccess) {
        throw std::runtime_error("CUDA kernel launch failed: " + std::string(hipGetErrorString(err)));
    }

    return output;
}


// scalar multiplication

__global__ void scaling_kernel(float* result, const float* matrix, float scalar, size_t rows, size_t cols)
{

    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row >= rows || col >= cols)
        return;

    int idx = row * cols + col;

    result[idx] = matrix[idx] * scalar;
}

DeviceMatrix DeviceMatrix::matScale(const DeviceMatrix& A, float scalar) {

    if (A.totalSize() == 0) {
        return A;
    }

    DeviceMatrix result(A.rows(), A.cols());

    dim3 dimBlock(TILE_WIDTH, TILE_HEIGHT);
    dim3 dimGrid((A.cols() + TILE_WIDTH - 1) / TILE_WIDTH, (A.rows() + TILE_HEIGHT - 1) / TILE_HEIGHT);

    scaling_kernel << <dimGrid, dimBlock >> > (result.device_matrix, A.device_matrix, scalar, A.rows(), A.cols());

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();

    if (err != hipSuccess) {
        throw std::runtime_error("CUDA kernel launch failed: " + std::string(hipGetErrorString(err)));
    }

    return result;
}

// sum-reduce trivial kernel

__global__ void sum_reduce_kernel(float* result, const float* matrix, size_t rows, size_t cols) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    float sum = 0.0f;

    if (row >= rows)
        return;

    for (size_t col = 0; col < cols; ++col) {
        sum += matrix[row * cols + col];
    }

    result[row] = sum;
}

DeviceMatrix DeviceMatrix::matColSum(const DeviceMatrix& A) {
    if (A.totalSize() == 0) {
        return A;
    }

    DeviceMatrix result(A.rows(), 1); // rezultat je vektor sa zbrojem svakog retka

    dim3 dimBlock(TILE_WIDTH, TILE_HEIGHT);
    dim3 dimGrid((result.cols() + TILE_WIDTH - 1) / TILE_WIDTH, (result.rows() + TILE_HEIGHT - 1) / TILE_HEIGHT);

    sum_reduce_kernel << <dimGrid, dimBlock >> > (result.device_matrix, A.device_matrix, A.rows(), A.cols());

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();

    if (err != hipSuccess) {
        throw std::runtime_error("CUDA kernel launch failed: " + std::string(hipGetErrorString(err)));
    }
    return result;
}


__inline__ __device__
float warp_reduce_sum(float val) {  // binary halving reduction


    unsigned mask = 0xffffffffu;  // mask of threads that participate in collective operation, ovdje sve pise : "https://developer.nvidia.com/blog/using-cuda-warp-level-primitives"


    val += __shfl_down_sync(mask, val, 16);
    val += __shfl_down_sync(mask, val, 8);
    val += __shfl_down_sync(mask, val, 4);
    val += __shfl_down_sync(mask, val, 2);
    val += __shfl_down_sync(mask, val, 1);
    return val;
}

__global__
void col_reduce_op(float* result, const float* matrix, size_t rows, size_t cols, size_t pitch_cols) {

    size_t row = blockIdx.y;
    if (row >= rows) return;

    extern __shared__ float s_warp_sums[];


    float local = 0.0f;
    for (size_t col = threadIdx.x; col < cols; col += blockDim.x) {   // matrica moze imati vise stupaca nego je alociranih threadova, threadovi coalesced iteriraju po svim segmentima u periodu velicine blockDim.x
        size_t idx = row * pitch_cols + col;
        local += matrix[idx];  
    } // u ovom trenutku svi threadovi u warpu imaju svoju vrijednost u registru local

    


    local = warp_reduce_sum(local);  // nakon ove naredbe prvi thread warpa ce imati u registru local vrijednost koja predstavlja sumu za warp


    int lane = threadIdx.x & 31;  // prvi thrad warpa
    int warpId = threadIdx.x >> 5;  // indeks warpa
    if (lane == 0) s_warp_sums[warpId] = local;  // prvi thread warpa upisuje vrijednost sume za warp u shared memoriju
    __syncthreads();  // svi warpovi bloka moraju doci do ove linije prije dalje

    // jedan blok == jedan row matrice
    float block_sum = 0.0f;  // sada treba napraviti redukciju s_warp_sums polja da bi dobili za cijeli blok
    int numWarps = (blockDim.x + 31) / 32;  // broj warpova bloka
    if (threadIdx.x < 32) {  // prvi thread svakog warpa racuna redukciju s_warp_sums polja
        float v = (threadIdx.x < numWarps) ? s_warp_sums[threadIdx.x] : 0.0f;  // svaki thread warpa koji racuna redukciju pristupa sumi jednog warpa, ako je thread izvan warpa, postavlja 0.0f
        block_sum = warp_reduce_sum(v); // redukcija
    }


    if (threadIdx.x == 0) result[row] = block_sum;  // u svaki redak matrice result se zapisuje vrijednost za blok
}


DeviceMatrix DeviceMatrix::matColSumV2(const DeviceMatrix& A) {

    if (A.totalSize() == 0 || A.cols() == 1)
        return A;


    DeviceMatrix result(A.rows(), 1);

    int threads = 256;
    dim3 block(threads, 1, 1);
    dim3 grid(1, A.rows(), 1);
    size_t shared_bytes = sizeof(float) * ((threads + 31) / 32);  // == broj warpova u bloku * sizeof(float); // ako ce se staticki alocirati onda najbolje 32 warpa jer (blockDim.x <= 1024 --> warps count <= 32) ali ovako je bolje dinamicki alocirati da ne bude bezveze previse shared memory alocirano

    col_reduce_op << <grid, block, shared_bytes >> > (result.device_matrix, A.device_matrix, A.rows(), A.cols(), A.cols());  // kasnije mozda bude trebalo pitched cols

    hipDeviceSynchronize();

    return result;

}


template<int ROWS_PER_TILE = 256>
__global__
void row_sum_op(float* result, const float* matrix, size_t rows, size_t cols) {

    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col >= cols)
        return;

    size_t row_start = blockIdx.y * ROWS_PER_TILE;
    size_t row_end = min(rows, row_start + ROWS_PER_TILE);

    float acc = 0.0f;

    for (size_t row = row_start; row < row_end; ++row) {
        acc += matrix[row * cols + col];
    }

    atomicAdd(&result[col], acc);

}

DeviceMatrix DeviceMatrix::matRowSumV2(const DeviceMatrix& A) {

    int threads = 256;
    const int rows_per_tile = 256;

    dim3 block(threads, 1, 1);
    dim3 grid((A.cols() + threads - 1) / threads,
              (A.rows() + rows_per_tile - 1 ) / rows_per_tile,
              1);

    DeviceMatrix result(1, A.cols());


    hipError_t err = hipGetLastError();
    
    row_sum_op<rows_per_tile> <<<grid, block >>> (result.device_matrix, A.device_matrix, A.rows(), A.cols());
    
    hipDeviceSynchronize();

    if (err != hipSuccess) {
        throw std::runtime_error("in procedure : matRowSumV2 : " + std::string(hipGetErrorString(err)));
    }

    return result;

}




// OPREZ OVO JE TRIVIJALNA IMPLEMENTACIJA, TREBA REIMPLEMENTIRATI S PAMETNIJIM ALGORITMOM KOJI KORISTI COALESCED MEM ACCESS ZA UBRZANJE

__global__ void row_reduce_kernel(float* result, const float* matrix, size_t rows, size_t cols) {

    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col >= cols)
        return;

    float sum = 0.0f;

    for (size_t row = 0; row < rows; ++row) {
        sum += matrix[row * cols + col];
    }

    result[col] = sum;
}

DeviceMatrix DeviceMatrix::matRowSum(const DeviceMatrix& A) {
    if (A.totalSize() == 0) {
        return A;
    }

    DeviceMatrix result(1, A.cols());

    dim3 dimBlock(TILE_WIDTH, TILE_HEIGHT);
    dim3 dimGrid((result.cols() + TILE_WIDTH - 1) / TILE_WIDTH, (result.rows() + TILE_HEIGHT - 1) / TILE_HEIGHT);

    row_reduce_kernel << < dimGrid, dimBlock >> > (result.device_matrix, A.device_matrix, A.rows(), A.cols());

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();

    if (err != hipSuccess) {
        throw std::runtime_error("CUDA kernel launch failed: " + std::string(hipGetErrorString(err)));
    }

    return result;
}


// Identity matrix

__global__ void identity_kernel(float* matrix, size_t size) {
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= size || col >= size) {
        return;
    }

    matrix[row * size + col] = (row == col) ? 1.0f : 0.0f;
}

DeviceMatrix DeviceMatrix::Identity(size_t size) {

    if (size == 0) {
        throw std::invalid_argument("Size must be greater than 0 for identity matrix.");
    }


    if (size < TILE_WIDTH || size < TILE_HEIGHT) {

        return DeviceMatrix(Matrix::Identity(size));
    }

    DeviceMatrix identity(size, size);

    dim3 dimBlock(TILE_WIDTH, TILE_HEIGHT);
    dim3 dimGrid((size + TILE_WIDTH - 1) / TILE_WIDTH, (size + TILE_WIDTH - 1) / TILE_WIDTH);

    identity_kernel << <dimGrid, dimBlock >> > (identity.device_matrix, size);

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();

    if (err != hipSuccess) {
        throw std::runtime_error("CUDA kernel launch failed: " + std::string(hipGetErrorString(err)));
    }

    return identity;
}


// Zero matrix

DeviceMatrix DeviceMatrix::Zero(size_t rows, size_t cols) {

    if (rows == 0 || cols == 0) {
        throw std::invalid_argument("Rows and columns must be greater than 0 for zero matrix.");
    }

    DeviceMatrix zero(rows, cols);

    return zero;
}




__global__ void random_matrix_kernel(float* matrix, size_t rows, size_t cols, float min_val, float max_val, unsigned long seed)
{
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    int idx = row * cols + col;

    if (row >= rows || col >= cols) return;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    float rand_val = hiprand_uniform(&state);

    matrix[idx] = min_val + rand_val * (max_val - min_val);

}


DeviceMatrix DeviceMatrix::Random(size_t rows, size_t cols, std::pair<float, float> range) {

    if (rows == 0 || cols == 0) {
        throw std::invalid_argument("Rows and columns must be greater than 0 for random matrix.");
    }

    /*if (rows < TILE_HEIGHT || cols < TILE_WIDTH) {
        return DeviceMatrix(Matrix::Random(rows, cols, range));
    }*/

    DeviceMatrix random_matrix(rows, cols);

    dim3 dimBlock(TILE_WIDTH, TILE_HEIGHT);
    dim3 dimGrid((cols + TILE_WIDTH - 1) / TILE_WIDTH, (rows + TILE_HEIGHT - 1) / TILE_HEIGHT);

    unsigned long seed = static_cast<unsigned long>(time(0));

    random_matrix_kernel <<<dimGrid, dimBlock >>> (random_matrix.device_matrix, rows, cols, range.first, range.second, seed);

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();

    if (err != hipSuccess) {
        throw std::runtime_error("CUDA kernel launch failed: " + std::string(hipGetErrorString(err)));
    }

    return random_matrix;
}


// BROADCASTING OPERATIONS

__global__ void broadcast_add_kernel(float* result, const float* matrix, const float* vector, size_t rows, size_t cols) {

    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;

    extern __shared__ float sh_vector[];

    if (row >= rows || col >= cols)
        return;


    if (threadIdx.x == 0) { // prvi stupac threadova u bloku ucitava cijeli stupac vektor u shared memoriju, prije je uvjet bilo col == 0, ali to je stvaralo problem jer samo prvi blok bi mogao ucitati vektor, a ostali blokovi bi bili prazni.
        sh_vector[row] = vector[row];
    }

    __syncthreads();

    float value = matrix[idx];
    float bias = sh_vector[row];

    result[idx] = value + bias;
}

DeviceMatrix DeviceMatrix::broadcastAdd(const DeviceMatrix& matrix, const DeviceMatrix& vector) {

    if (vector.cols() != 1 || vector.rows() != matrix.rows()) {
        throw std::runtime_error("Vector must be a single row with the same number of columns as the matrix.");
    }

    DeviceMatrix result(matrix.rows(), matrix.cols());

    dim3 dimBlock(TILE_WIDTH, TILE_HEIGHT);
    dim3 dimGrid((matrix.cols() + TILE_WIDTH - 1) / TILE_WIDTH, (matrix.rows() + TILE_HEIGHT - 1) / TILE_HEIGHT);

    size_t shared_memory_size = vector.rows() * sizeof(float);

    broadcast_add_kernel << <dimGrid, dimBlock, shared_memory_size >> > (result.device_matrix, matrix.device_matrix, vector.device_matrix, matrix.rows(), matrix.cols());

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();

    if (err != hipSuccess) {
        throw std::runtime_error("CUDA kernel launch failed: " + std::string(hipGetErrorString(err)));
    }

    return result;
}


// ReLU-aktivacija

__global__ void ReLU_kernel(float* output, const float* input, size_t rows, size_t cols) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row >= rows || col >= cols) {
        return;
    }

    int idx = row * cols + col;

    output[idx] = (input[idx] > 0) ? input[idx] : 0;
}


DeviceMatrix DeviceMatrix::ReLU(const DeviceMatrix& input) {

    if (input.totalSize() == 0) {
        throw std::runtime_error("ReLU forward error: Input matrix is empty");
    }

    DeviceMatrix output(input.rows(), input.cols());

    dim3 blockSize(TILE_WIDTH, TILE_HEIGHT);
    dim3 gridSize((input.cols() + TILE_WIDTH - 1) / TILE_WIDTH, (input.rows() + TILE_HEIGHT - 1) / TILE_HEIGHT);

    ReLU_kernel << <gridSize, blockSize >> > (output.device_matrix, input.device_matrix, input.rows(), input.cols());

    hipDeviceSynchronize();

    hipError_t cuda_error = hipGetLastError();

    if (cuda_error != hipSuccess) {
        throw std::runtime_error("ReLU forward error: " + std::string(hipGetErrorString(cuda_error)));
    }

    return output;
}

__global__ void ReLU_gradient_kernel(float* output, const float* relu_output, size_t rows, size_t cols) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row >= rows || col >= cols) {
        return;
    }

    int idx = row * cols + col;
    output[idx] = (relu_output[idx] > 0) ? 1.0f : 0.0f;
}

DeviceMatrix DeviceMatrix::ReLUGradient(const DeviceMatrix& relu_output) {

    if (relu_output.totalSize() == 0) {
        throw std::runtime_error("ReLU gradient error: Input matrix is empty");
    }

    DeviceMatrix output(relu_output.rows(), relu_output.cols());

    dim3 blockSize(TILE_WIDTH, TILE_HEIGHT);
    dim3 gridSize((relu_output.cols() + TILE_WIDTH - 1) / TILE_WIDTH, (relu_output.rows() + TILE_HEIGHT - 1) / TILE_HEIGHT);

    ReLU_gradient_kernel << <gridSize, blockSize >> > (output.device_matrix, relu_output.device_matrix, relu_output.rows(), relu_output.cols());

    hipDeviceSynchronize();

    hipError_t cuda_error = hipGetLastError();


    if (cuda_error != hipSuccess) {
        throw std::runtime_error("ReLU gradient error: " + std::string(hipGetErrorString(cuda_error)));
    }

    return output;
}

// sigmoid activation

__global__ void sigmoid_kernel(float* output, const float* input, size_t rows, size_t cols) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row >= rows || col >= cols)
        return;

    int idx = row * cols + col;

    output[idx] = 1.0f / (1.0f + expf(-input[idx]));
}


DeviceMatrix DeviceMatrix::Sigmoid(const DeviceMatrix& input) {

    if (input.totalSize() == 0) {
        throw std::runtime_error("Sigmoid forward error: Input matrix is empty");
    }

    DeviceMatrix output(input.rows(), input.cols());

    dim3 blockSize(TILE_WIDTH, TILE_HEIGHT);
    dim3 gridSize((input.cols() + TILE_WIDTH - 1) / TILE_WIDTH, (input.rows() + TILE_HEIGHT - 1) / TILE_HEIGHT);

    sigmoid_kernel << <gridSize, blockSize >> > (output.device_matrix, input.device_matrix, input.rows(), input.cols());

    hipDeviceSynchronize();

    hipError_t cuda_error = hipGetLastError();

    if (cuda_error != hipSuccess) {
        throw std::runtime_error("Sigmoid forward error: " + std::string(hipGetErrorString(cuda_error)));
    }

    return output;
}

__global__ void sigmoid_gradient_kernel(float* gradient, const float* sigmoid_output, size_t rows, size_t cols) {

    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row >= rows || col >= cols)
        return;

    int idx = row * cols + col;

    float sigma = sigmoid_output[idx];

    gradient[idx] = sigma * (1.0f - sigma);
}

DeviceMatrix DeviceMatrix::SigmoidGradient(const DeviceMatrix& sigmoid_output) {

    if (sigmoid_output.totalSize() == 0) {
        throw std::runtime_error("Sigmoid gradient error: Input matrix or sigmoid value matrix is empty");
    }

    DeviceMatrix output(sigmoid_output.rows(), sigmoid_output.cols());

    dim3 blockSize(TILE_WIDTH, TILE_HEIGHT);
    dim3 gridSize((sigmoid_output.cols() + TILE_WIDTH - 1) / TILE_WIDTH, (sigmoid_output.rows() + TILE_HEIGHT - 1) / TILE_HEIGHT);

    sigmoid_gradient_kernel << <gridSize, blockSize >> > (output.device_matrix, sigmoid_output.device_matrix, sigmoid_output.rows(), sigmoid_output.cols());

    hipDeviceSynchronize();

    hipError_t cuda_error = hipGetLastError();

    if (cuda_error != hipSuccess) {
        throw std::runtime_error("Sigmoid gradient error: " + std::string(hipGetErrorString(cuda_error)));
    }

    return output;
}

// mora se implementirati jos


__global__ void MSE_kernel(float* output, const float* target, size_t rows, size_t cols) {}

// POZOR, MSE TRENUTNO U TRIVIJALNOJ IMPLEMENTACIJI, SEMANTICKI ISPRAVAN, ALI NIKAKO OPTIMALAN PO PITANJU VREMENA

DeviceMatrix DeviceMatrix::MSE(const DeviceMatrix& output, const DeviceMatrix& target) {

    DeviceMatrix result = DeviceMatrix::matSub(output, target);

    result = DeviceMatrix::matElementWiseMul(result, result);

    result = DeviceMatrix::matRowSum(result);

    result = DeviceMatrix::matColSum(result);

    result = DeviceMatrix::matScale(result, 1.0f / (output.rows() * output.cols()));

    return result;
}


__global__ void MSE_gradient_kernel(float* gradient, const float* output, const float* target, size_t rows, size_t cols) {

    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    if (row >= rows || col >= cols)
        return;

    int idx = row * cols + col;

    gradient[idx] = 2.0f * (output[idx] - target[idx]) / (rows * cols);
}

// TRIVIJALNA IMPLEMENTACIJA, SEMANTICKI ISPRAVAN, ALI NIKAKO OPTIMALAN PO PITANJU VREMENA

DeviceMatrix DeviceMatrix::MSEGradient(const DeviceMatrix& output, const DeviceMatrix& target) {

    if (output.totalSize() == 0 || target.totalSize() == 0) {
        throw std::runtime_error("MSE gradient error: Output or target matrix is empty");
    }

    if (output.rows() != target.rows() || output.cols() != target.cols()) {
        throw std::runtime_error("MSE gradient error: Output and target matrices must have the same dimensions");
    }

    DeviceMatrix gradient(output.rows(), output.cols());

    dim3 blockSize(TILE_WIDTH, TILE_HEIGHT);
    dim3 gridSize((output.cols() + TILE_WIDTH - 1) / TILE_WIDTH, (output.rows() + TILE_HEIGHT - 1) / TILE_HEIGHT);

    MSE_gradient_kernel << <gridSize, blockSize >> > (gradient.device_matrix, output.device_matrix, target.device_matrix, output.rows(), output.cols());

    hipDeviceSynchronize();

    hipError_t cuda_error = hipGetLastError();

    if (cuda_error != hipSuccess) {
        throw std::runtime_error("MSE gradient error: " + std::string(hipGetErrorString(cuda_error)));
    }

    return gradient;
}

__global__ void BCE_gradient_kernel(float* gradient, const float* output, const float* target, size_t rows, size_t cols) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    if (row >= rows || col >= cols)
        return;

    int idx = row * cols + col;

    float y_hat = output[idx];
    float y = target[idx];

    // epsilon da izbjegnemo log(0) i dijeljenje s 0
    const float eps = 1e-8f;
    gradient[idx] = (y_hat - y) / (max(y_hat * (1.0f - y_hat), eps));
}

DeviceMatrix DeviceMatrix::BCEGradient(const DeviceMatrix& output, const DeviceMatrix& target) {
    if (output.totalSize() == 0 || target.totalSize() == 0)
        throw std::runtime_error("BCE gradient: matrices empty");

    if (output.rows() != target.rows() || output.cols() != target.cols())
        throw std::runtime_error("BCE gradient: size mismatch");

    DeviceMatrix gradient(output.rows(), output.cols());

    dim3 blockSize(TILE_WIDTH, TILE_HEIGHT);
    dim3 gridSize((output.cols() + TILE_WIDTH - 1) / TILE_WIDTH,
        (output.rows() + TILE_HEIGHT - 1) / TILE_HEIGHT);

    BCE_gradient_kernel << <gridSize, blockSize >> > (
        gradient.device_matrix, output.device_matrix, target.device_matrix,
        output.rows(), output.cols()
        );

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        throw std::runtime_error("BCE gradient kernel error: " + std::string(hipGetErrorString(err)));

    return gradient;
}

__global__ void BCE_kernel(const float* output, const float* target, float* result, size_t rows, size_t cols) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    if (row >= rows || col >= cols) return;

    int idx = row * cols + col;
    const float eps = 1e-8f; // stabilnost loga
    float y_hat = output[idx];
    float y = target[idx];

    result[idx] = -(y * logf(fmaxf(y_hat, eps)) + (1.0f - y) * logf(fmaxf(1.0f - y_hat, eps)));
}

DeviceMatrix DeviceMatrix::BCE(const DeviceMatrix& output, const DeviceMatrix& target) {
    if (output.totalSize() == 0 || target.totalSize() == 0)
        throw std::runtime_error("BCE: matrices empty");

    if (output.rows() != target.rows() || output.cols() != target.cols())
        throw std::runtime_error("BCE: size mismatch");


    DeviceMatrix bce_values(output.rows(), output.cols());

    dim3 blockSize(TILE_WIDTH, TILE_HEIGHT);
    dim3 gridSize((output.cols() + TILE_WIDTH - 1) / TILE_WIDTH,
        (output.rows() + TILE_HEIGHT - 1) / TILE_HEIGHT);

    BCE_kernel << <gridSize, blockSize >> > (
        output.device_matrix,
        target.device_matrix,
        bce_values.device_matrix,
        output.rows(), output.cols()
        );

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        throw std::runtime_error("BCE kernel error: " + std::string(hipGetErrorString(err)));

 
    DeviceMatrix row_sum = DeviceMatrix::matRowSum(bce_values);
    DeviceMatrix total_sum = DeviceMatrix::matColSum(row_sum); 


    float scale = 1.0f / static_cast<float>(output.totalSize());
    DeviceMatrix mean_bce = DeviceMatrix::matScale(total_sum, scale);

    return mean_bce; 
}





};